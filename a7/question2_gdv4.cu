#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <iostream>

#define N 500000000

using namespace std;

void warmUpGPU();
__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C);

int main(int argc, char *argv[])
{

	warmUpGPU();


	unsigned int * A;
	unsigned int * B;
	unsigned int * C;
	unsigned int * C_CPU;

	A=(unsigned int *)malloc(sizeof(unsigned int)*N);
	B=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C_CPU=(unsigned int *)malloc(sizeof(unsigned int)*N);


	printf("\nSize of A+B+C (GiB): %f",(sizeof(unsigned int)*N*3.0)/(1024.0*1024.0*1024.0));


	//init:
	int i=0;
	for (i=0; i<N; i++){
		A[i]=i;
		B[i]=i;
		C[i]=0;
		C_CPU[i]=0;
	}

   double start_time = omp_get_wtime();
	//CPU version:


	for (int i=0; i<N; i++){
		C_CPU[i]=A[i]+B[i];
	}

   double cpu_end = omp_get_wtime();

	//CUDA error code:
	hipError_t errCode=hipSuccess;

	unsigned int * dev_A;
	unsigned int * dev_B;
	unsigned int * dev_C;

	//allocate on the device: A, B, C
   double start_mem_to = omp_get_wtime();
	errCode=hipMalloc((unsigned int**)&dev_A, sizeof(unsigned int)*N);
	if(errCode != hipSuccess) {
	cout << "\nError: A error with code " << errCode << endl;
	}

	errCode=hipMalloc((unsigned int**)&dev_B, sizeof(unsigned int)*N);
	if(errCode != hipSuccess) {
	cout << "\nError: B error with code " << errCode << endl;
	}

	errCode=hipMalloc((unsigned int**)&dev_C, sizeof(unsigned int)*N);
	if(errCode != hipSuccess) {
	cout << "\nError: C error with code " << errCode << endl;
	}

	//copy A to device
	errCode=hipMemcpy( dev_A, A, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl;
	}

	//copy B to device
	errCode=hipMemcpy( dev_B, B, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl;
	}

	//copy C to device (initialized to 0)
	errCode=hipMemcpy( dev_C, C, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl;
	}
   double end_mem_to = omp_get_wtime(); //Also used for calculating gpu time
	//execute kernel
	const unsigned int totalBlocks=ceil(N*1.0/1024.0);
	printf("\ntotal blocks: %d",totalBlocks);
	vectorAdd<<<totalBlocks,1024>>>(dev_A, dev_B, dev_C);

	if(errCode != hipSuccess){
		cout<<"Error after kernel launch "<<errCode<<endl;
	}
   double start_mem_from = omp_get_wtime(); //Also used for calculating gpu time
	//copy data from device to host
	errCode=hipMemcpy( C, dev_C, sizeof(unsigned int)*N, hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	cout << "\nError: getting C result form GPU error with code " << errCode << endl;
	}
   double end_mem_from = omp_get_wtime(); //Also used to calc total time

   printf("Time to complete CPU: %f\n",cpu_end-start_time);
   printf("Time to complete GPU: %f\n",start_mem_from-end_mem_to);
   printf("Time to transfer data: %f\n",end_mem_from-start_mem_from+end_mem_to-end_mem_from);
   printf("Time spent in Kernal: %f\n",end_mem_from-start_time);
	return 0;
}

__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C) {

unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x);

if (tid>=N){
	return;
}
C[tid]=A[tid]+B[tid];

return;
}



__global__ void warmup(unsigned int * tmp) {
if (threadIdx.x==0)
*tmp=555;

return;
}



void warmUpGPU(){


printf("\nWarming up GPU for time trialing...\n");
unsigned int * dev_tmp;
unsigned int * tmp;
tmp=(unsigned int*)malloc(sizeof(unsigned int));
*tmp=0;
hipError_t errCode=hipSuccess;
errCode=hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));
	if(errCode != hipSuccess) {
	cout << "\nError: dev_tmp error with code " << errCode << endl;
	}

warmup<<<1,256>>>(dev_tmp);

//copy data from device to host
	errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	cout << "\nError: getting tmp result form GPU error with code " << errCode << endl;
	}

	hipDeviceSynchronize();

	printf("\ntmp (changed to 555 on GPU): %d",*tmp);

hipFree(dev_tmp);

return;
}
